#include "hip/hip_runtime.h"

#include "cuda_mfields.h"

#include "fields.hxx"

#include <cmath>

enum
{ // FIXME, duplicated
  JXI,
  JYI,
  JZI,
  EX,
  EY,
  EZ,
  HX,
  HY,
  HZ,
  NR_FIELDS,
};

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <mrc_profile.h>

struct prof_globals prof_globals; // FIXME

int prof_register(const char* name, float simd, int flops, int bytes)
{
  return 0;
}

// ----------------------------------------------------------------------
// init_wave

double init_wave(double x, double y, int m)
{
  double kx = 2. * M_PI / 80., ky = 2. * M_PI / 80.;
  switch (m) {
    case EX: return 1. / sqrtf(2.) * sin(kx * x + ky * y);
    case EY: return -1. / sqrtf(2.) * sin(kx * x + ky * y);
    case HZ: return sin(kx * x + ky * y);
    default: return 0.;
  }
}

// ----------------------------------------------------------------------
// main

int main(void)
{
  using fields_t = fields3d<float>;
  using real_t = fields_t::real_t;

  Grid_t grid{};
  grid.ldims = {1, 8, 8};
  grid.dx = {1., 10., 10.};

  Grid_t::Patch patch{};
  patch.xb = {0., 0., 0.};
  grid.patches.push_back(patch);

  int n_fields = 9;
  Int3 ibn = {0, 2, 2};
  struct cuda_mfields* cmflds = new cuda_mfields(grid, n_fields, ibn);

  int n_patches = cmflds->n_patches;
  const Vec3<double>& dx = grid.dx;

  for (int m = 0; m < n_fields; m++) {
    cmflds->zero_comp_yz(m);
  }

  cmflds->dump("cmflds.json");

  auto mflds = hostMirror(*cmflds);
  auto flds = mflds[0];
  for (int p = 0; p < n_patches; p++) {
    for (int k = flds.ib[2]; k < flds.ib[2] + flds.im[2]; k++) {
      for (int j = flds.ib[1]; j < flds.ib[1] + flds.im[1]; j++) {
        for (int i = flds.ib[0]; i < flds.ib[0] + flds.im[0]; i++) {
          real_t x_cc = (i + .5) * dx[0];
          real_t y_cc = (j + .5) * dx[1];
          real_t x_nc = i * dx[0];
          real_t y_nc = j * dx[1];
          flds(EX, i, j, k) = init_wave(x_cc, y_nc, EX);
          flds(EY, i, j, k) = init_wave(x_nc, y_cc, EY);
          flds(EZ, i, j, k) = init_wave(x_nc, y_nc, EZ);
          flds(HX, i, j, k) = init_wave(x_nc, y_cc, HX);
          flds(HY, i, j, k) = init_wave(x_cc, y_nc, HY);
          flds(HZ, i, j, k) = init_wave(x_cc, y_cc, HZ);
        }
      }
    }
  }
  copy(mflds, *cmflds);

  cmflds->dump("cmflds_wave.json");

  float dt = dx[1];
  cuda_push_fields_E_yz(cmflds, .5 * dt);
  cuda_push_fields_H_yz(cmflds, dt);
  cuda_push_fields_E_yz(cmflds, .5 * dt);

  cmflds->dump("cmflds_wave_1.json");

  delete cmflds;
}
