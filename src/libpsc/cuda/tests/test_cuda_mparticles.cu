#include "hip/hip_runtime.h"

#include "cuda_mparticles.cuh"
#include "cuda_mparticles_sort.cuh"
#include "cuda_collision.cuh"
#include "cuda_test.hxx"
#include "bs.hxx"

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <mrc_profile.h>

#include "gtest/gtest.h"

struct prof_globals prof_globals; // FIXME

int prof_register(const char* name, float simd, int flops, int bytes)
{
  return 0;
}

using CudaMparticles = cuda_mparticles<BS144>;

// ----------------------------------------------------------------------
// cuda_mparticles_add_particles_test_1
//
// add 1 particle at the center of each cell, in the "wrong" order in each
// patch (C order, but to get them ordered by block, they need to be reordered
// into Fortran order, a.k.a., this will exercise the initial sorting

void cuda_mparticles_add_particles_test_1(CudaMparticles& cmprts,
                                          std::vector<uint>& n_prts_by_patch)
{
  using Particle = CudaMparticles::Particle;
  using real_t = Particle::real_t;
  using Real3 = Particle::Real3;

  const Grid_t& grid = cmprts.grid_;
  Int3 ldims = grid.ldims;

  uint n_prts = 0;
  for (int p = 0; p < cmprts.n_patches(); p++) {
    n_prts_by_patch[p] = ldims[0] * ldims[1] * ldims[2];
    n_prts += n_prts_by_patch[p];
  }

  auto dx = grid.domain.dx;

  std::vector<Particle> buf;
  buf.reserve(n_prts);

  for (int p = 0; p < grid.n_patches(); p++) {
    for (int i = 0; i < ldims[0]; i++) {
      for (int j = 0; j < ldims[1]; j++) {
        for (int k = 0; k < ldims[2]; k++) {
          buf.push_back(
            Particle{{real_t(dx[0] * (i + .5f)), real_t(dx[1] * (j + .5f)),
                      real_t(dx[2] * (k + .5f))},
                     {real_t(i), real_t(j), real_t(k)},
                     1.,
                     0});
        }
      }
    }
  }
  cmprts.inject_initial(buf, n_prts_by_patch);
}

// ======================================================================
// CudaMparticlesTest

struct CudaMparticlesTest
  : TestBase<CudaMparticles>
  , ::testing::Test
{
  std::unique_ptr<Grid_t> grid_;

  void SetUp()
  {
    auto domain = Grid_t::Domain{{1, 8, 4}, {1., 80., 40.}};
    auto bc = psc::grid::BC{};
    auto kinds = Grid_t::Kinds{};
    auto norm = Grid_t::Normalization{};
    double dt = .1;
    grid_.reset(new Grid_t{domain, bc, kinds, norm, dt});
  }
};

// ----------------------------------------------------------------------
TEST_F(CudaMparticlesTest, ConstructorDestructor)
{
  grid_->kinds.push_back(Grid_t::Kind(-1., 1., "electron"));
  grid_->kinds.push_back(Grid_t::Kind(1., 25., "ion"));
  auto cmprts = CudaMparticles{*grid_};
  EXPECT_EQ(cmprts.n_patches(), 1);
}

// ----------------------------------------------------------------------
TEST_F(CudaMparticlesTest, SetParticles)
{
  grid_->kinds.push_back(Grid_t::Kind(-1., 1., "electron"));
  grid_->kinds.push_back(Grid_t::Kind(1., 25., "ion"));
  auto cmprts = CudaMparticles{*grid_};

  std::vector<uint> n_prts_by_patch(cmprts.n_patches());
  cuda_mparticles_add_particles_test_1(cmprts, n_prts_by_patch);

  // check that particles are in C order
  int n = 0;
  auto accessor = cmprts.accessor();
  for (auto prt : accessor[0]) {
    int nn = n++;
    int k = nn % grid_->ldims[2];
    nn /= grid_->ldims[2];
    int j = nn % grid_->ldims[1];
    nn /= grid_->ldims[1];
    int i = nn;
    EXPECT_FLOAT_EQ(prt.x()[0], (i + .5) * grid_->domain.dx[0]);
    EXPECT_FLOAT_EQ(prt.x()[1], (j + .5) * grid_->domain.dx[1]);
    EXPECT_FLOAT_EQ(prt.x()[2], (k + .5) * grid_->domain.dx[2]);
  }
}

// ---------------------------------------------------------------------
// SetupInternalsDetail
//
// Tests the pieces that go into setup_internals()

TEST_F(CudaMparticlesTest, SetupInternalsDetail)
{
  grid_->kinds.push_back(Grid_t::Kind(-1., 1., "electron"));
  grid_->kinds.push_back(Grid_t::Kind(1., 25., "ion"));

  std::vector<Particle> prts = {
    {{.5, 75., 15.}, {}, 0., 0},
    {{.5, 35., 15.}, {}, 0., 0},
    {{.5, 5., 5.}, {}, 0., 0},
  };

  // can't use make_cmprts() from vector here, since that'll sort etc
  auto cmprts = CudaMparticles{*grid_};
  cmprts.inject_initial(prts, {uint(prts.size())});

  auto& d_id = cmprts.by_block_.d_id;
  auto& d_bidx = cmprts.by_block_.d_idx;
  EXPECT_EQ(d_bidx[0], 0);
  EXPECT_EQ(d_bidx[1], 0);
  EXPECT_EQ(d_bidx[2], 0);
  EXPECT_EQ(d_id[0], 0);
  EXPECT_EQ(d_id[1], 0);
  EXPECT_EQ(d_id[2], 0);

  EXPECT_TRUE(cmprts.check_in_patch_unordered_slow());
  cmprts.by_block_.find_indices_ids(cmprts);

  EXPECT_EQ(d_bidx[0], 1);
  EXPECT_EQ(d_bidx[1], 0);
  EXPECT_EQ(d_bidx[2], 0);
  EXPECT_EQ(d_id[0], 0);
  EXPECT_EQ(d_id[1], 1);
  EXPECT_EQ(d_id[2], 2);

  EXPECT_TRUE(cmprts.check_bidx_id_unordered_slow());
  cmprts.by_block_.stable_sort();

  EXPECT_EQ(d_bidx[0], 0);
  EXPECT_EQ(d_bidx[1], 0);
  EXPECT_EQ(d_bidx[2], 1);
  EXPECT_EQ(d_id[0], 1);
  EXPECT_EQ(d_id[1], 2);
  EXPECT_EQ(d_id[2], 0);

  cmprts.by_block_.reorder_and_offsets(cmprts);

  float4 xi4_0 = cmprts.storage.xi4[0], xi4_1 = cmprts.storage.xi4[1],
         xi4_2 = cmprts.storage.xi4[2];
  EXPECT_FLOAT_EQ(xi4_0.y, 35.);
  EXPECT_FLOAT_EQ(xi4_0.z, 15.);
  EXPECT_FLOAT_EQ(xi4_1.y, 5.);
  EXPECT_FLOAT_EQ(xi4_1.z, 5.);
  EXPECT_FLOAT_EQ(xi4_2.y, 75.);
  EXPECT_FLOAT_EQ(xi4_2.z, 15.);

  auto& d_off = cmprts.by_block_.d_off;
  EXPECT_EQ(d_off[0], 0);
  EXPECT_EQ(d_off[1], 2);
  EXPECT_EQ(d_off[2], 3);

  EXPECT_TRUE(cmprts.check_ordered());
}

// ---------------------------------------------------------------------
// SortByCellDetail
//
// Tests the pieces that go into setup_internals()

TEST_F(CudaMparticlesTest, SortByCellDetail)
{
  grid_->kinds.push_back(Grid_t::Kind(-1., 1., "electron"));
  grid_->kinds.push_back(Grid_t::Kind(1., 25., "ion"));

  std::vector<Particle> prts = {
    {{.5, 75., 15.}, {}, 0., 0},
    {{.5, 35., 15.}, {}, 0., 0},
    {{.5, 5., 5.}, {}, 0., 0},
  };

  // can't use make_cmprts() from vector here, since that'll sort etc
  auto cmprts = CudaMparticles{*grid_};
  cmprts.inject_initial(prts, {uint(prts.size())});
  EXPECT_TRUE(cmprts.check_in_patch_unordered_slow());

  auto sort_by_cell = cuda_mparticles_sort{cmprts.n_cells()};
  auto& d_idx = sort_by_cell.d_idx;
  auto& d_id = sort_by_cell.d_id;

  sort_by_cell.find_indices_ids(cmprts);
  EXPECT_EQ(d_idx[0], 15);
  EXPECT_EQ(d_idx[1], 11);
  EXPECT_EQ(d_idx[2], 0);
  EXPECT_EQ(d_id[0], 0);
  EXPECT_EQ(d_id[1], 1);
  EXPECT_EQ(d_id[2], 2);

  sort_by_cell.stable_sort_cidx();
  EXPECT_EQ(d_idx[0], 0);
  EXPECT_EQ(d_idx[1], 11);
  EXPECT_EQ(d_idx[2], 15);
  EXPECT_EQ(d_id[0], 2);
  EXPECT_EQ(d_id[1], 1);
  EXPECT_EQ(d_id[2], 0);

  sort_by_cell.find_offsets();
  auto& d_off = sort_by_cell.d_off;
  EXPECT_EQ(d_off[0], 0);
  for (int c = 1; c <= 11; c++) {
    EXPECT_EQ(d_off[c], 1) << "c = " << c;
  }
  for (int c = 12; c <= 15; c++) {
    EXPECT_EQ(d_off[c], 2) << "c = " << c;
  }
  EXPECT_EQ(d_off[16], 3);

  sort_by_cell.reorder(cmprts);
  float4 xi4_0 = cmprts.storage.xi4[0], xi4_1 = cmprts.storage.xi4[1],
         xi4_2 = cmprts.storage.xi4[2];
  EXPECT_FLOAT_EQ(xi4_0.y, 5.);
  EXPECT_FLOAT_EQ(xi4_0.z, 5.);
  EXPECT_FLOAT_EQ(xi4_1.y, 35.);
  EXPECT_FLOAT_EQ(xi4_1.z, 15.);
  EXPECT_FLOAT_EQ(xi4_2.y, 75.);
  EXPECT_FLOAT_EQ(xi4_2.z, 15.);
}

// ----------------------------------------------------------------------
// SetupInternals
//
// tests setup_internals() itself, on a slightly bigger set of particles

TEST_F(CudaMparticlesTest, SetupInternals)
{
  grid_->kinds.push_back(Grid_t::Kind(1., 1., "test species"));
  auto cmprts = CudaMparticles{*grid_};

  std::vector<uint> n_prts_by_patch(cmprts.n_patches());
  cuda_mparticles_add_particles_test_1(cmprts, n_prts_by_patch);

  cmprts.check_in_patch_unordered_slow();

  cmprts.setup_internals();

  // check that particles are now in Fortran order
  int cur_bidx = 0;
  auto accessor = cmprts.accessor();
  for (auto prt : accessor[0]) {
    float4 xi = {prt.x()[0], prt.x()[1], prt.x()[2]};
    int bidx = cmprts.blockIndex(xi, 0);
    EXPECT_GE(bidx, cur_bidx);
    cur_bidx = bidx;
  }

  cmprts.check_ordered();
}

// ----------------------------------------------------------------------
// CudaCollision

TEST_F(CudaMparticlesTest, CudaCollision)
{
  grid_->kinds.push_back(Grid_t::Kind(1., 1., "test species"));

  std::vector<Particle> prts = {
    {{.5, 75., 15.}, {1.0, 0., 0.}, 1., 0},
    {{.5, 75., 15.}, {1.1, 0., 0.}, 1., 0},
    {{.5, 75., 15.}, {1.2, 0., 0.}, 1., 0},
    {{.5, 35., 5.}, {0., 1.0, 0.}, 1., 0},
    {{.5, 35., 5.}, {0., 1.1, 0.}, 1., 0},
    {{.5, 35., 5.}, {0., 1.2, 0.}, 1., 0},
    {{.5, 35., 5.}, {0., 1.3, 0.}, 1., 0},
    {{.5, 35., 5.}, {0., 1.4, 0.}, 1., 0},
    {{.5, 5., 5.}, {0., 0., 1.0}, 1., 0},
    {{.5, 5., 5.}, {0., 0., 1.1}, 1., 0},
  };

  auto cmprts = CudaMparticles{*grid_};
  cmprts.injector()[0].raw(prts);

  cmprts.check_ordered();

  int interval = 1;
  double nu = .1;
  int nicell = 10;
  double dt = .1;
  CudaCollision<CudaMparticles, RngStateCuda> coll{interval, nu, nicell, dt};

  coll(cmprts);
  auto accessor = cmprts.accessor();
  for (auto prt : accessor[0]) {
    printf("xi %g %g pxi %g %g %g\n", prt.x()[1], prt.x()[2], prt.u()[0],
           prt.u()[1], prt.u()[2]);
  }
}

// ======================================================================
// main

int main(int argc, char** argv)
{
  MPI_Init(&argc, &argv);

  ::testing::InitGoogleTest(&argc, argv);
  int rc = RUN_ALL_TESTS();

  MPI_Finalize();
  return rc;
}
