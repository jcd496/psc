#include "hip/hip_runtime.h"

#include "cuda_mparticles.cuh"
#include "cuda_bndp.h"
#include "cuda_test.hxx"

#include <mrc_profile.h>

#include "gtest/gtest.h"

struct prof_globals prof_globals; // FIXME

int prof_register(const char* name, float simd, int flops, int bytes)
{
  return 0;
}

using CudaMparticles = cuda_mparticles<BS144>;

// ======================================================================
// CudaMparticlesBndTest

struct CudaMparticlesBndTest
  : TestBase<CudaMparticles>
  , ::testing::Test
{
  using Double3 = Vec3<double>;

  std::unique_ptr<Grid_t> grid;
  std::unique_ptr<CudaMparticles> cmprts_;
  std::unique_ptr<cuda_bndp<CudaMparticles, dim_yz>> cbndp;

  void SetUp()
  {
    auto domain =
      Grid_t::Domain{{1, 32, 32}, {1., 320., 320.}, {0., 0., 0.}, {1, 2, 2}};
    auto bc = psc::grid::BC{};
    auto kinds =
      Grid_t::Kinds{Grid_t::Kind{1., 1., "k0"}, Grid_t::Kind{1., 1., "k1"},
                    Grid_t::Kind{1., 1., "k2"}, Grid_t::Kind{1., 1., "k3"}};
    auto norm = Grid_t::Normalization{};
    double dt = .1;
    grid.reset(new Grid_t(domain, bc, kinds, norm, dt));

    cmprts_.reset(new CudaMparticles{*grid});
    auto& cmprts = *cmprts_;

    // (ab)use kind to track particle more easily in the test
    {
      auto inj = cmprts.injector();
      // patch 0
      inj[0](psc::particle::Inject{{.5, 35., 5.}, {}, 0., 0});
      inj[0](psc::particle::Inject{{.5, 155., 5.}, {}, 0., 1});
      // patch 1
      inj[1](psc::particle::Inject{{.5, 195., 5.}, {}, 0., 2});
      inj[1](psc::particle::Inject{{.5, 315., 5.}, {}, 0., 3});
    }

    // move every particle one full cell to the right (+y, that is)
    // (position doesn't actually matter since we'll only look at bidx)
    for (int n = 0; n < cmprts.n_prts; n++) {
      auto prt = cmprts.storage.load(n);
      prt.x()[1] += 10.;
      cmprts.storage.store(prt, n);
    }
    auto& d_bidx = cmprts.by_block_.d_idx;
    d_bidx[0] = 0 + 1 * 3; // +1 in y, 0 in z
    d_bidx[1] = CUDA_BND_S_OOB;
    d_bidx[2] = 0 + 1 * 3; // +1 in y, 0 in z
    d_bidx[3] = CUDA_BND_S_OOB;

#if 0
    cmprts.dump();
#endif

    cbndp.reset(new cuda_bndp<cuda_mparticles<BS144>, dim_yz>(*grid));
  }
};

// ----------------------------------------------------------------------
// BndPrep
//
// tests cuda_bndp::prep()

TEST_F(CudaMparticlesBndTest, BndPrep)
{
  cbndp->prep(cmprts_.get());

  // particles 0 and 2 remain in their patch,
  // particles 1 and 3 leave their patch and need special handling
  EXPECT_EQ(cbndp->bufs[0].size(), 1);
  EXPECT_EQ(cbndp->bufs[1].size(), 1);
  EXPECT_EQ(cbndp->bufs[0][0].kind, 1);
  EXPECT_EQ(cbndp->bufs[1][0].kind, 3);
}

// ----------------------------------------------------------------------
// BndPrepDetail
//
// tests the pieces that go into cuda_bndp::prep()

TEST_F(CudaMparticlesBndTest, BndPrepDetail)
{
  auto& cmprts = *cmprts_;

  // test spine_reduce
  cbndp->spine_reduce(cmprts_.get());

#if 0
  for (int b = 0; b < cmprts.n_blocks; b++) {
    printf("b %d:", b);
    for (int n = 0; n < 10; n++) {
      int cnt = cbndp->d_spine_cnts[10*b + n];
      printf(" %3d", cnt);
    }
    printf("\n");
  }
#endif

  for (int b = 0; b < cmprts.n_blocks; b++) {
    for (int n = 0; n < 10; n++) {
      int cnt = cbndp->d_spine_cnts[10 * b + n];
      // one particle each moves to block 1, 17, respectively, from the left
      // (-y: 3)
      if ((b == 1 && n == 3) || (b == 17 && n == 3)) {
        EXPECT_EQ(cnt, 1) << "where b = " << b << " n = " << n;
      } else {
        EXPECT_EQ(cnt, 0) << "where b = " << b << " n = " << n;
      }
    }
  }

#if 0
  printf("oob: ");
  for (int b = 0; b < cmprts.n_blocks + 1; b++) {
    int cnt = cbndp->d_spine_cnts[10*cmprts.n_blocks + b];
    printf(" %3d", cnt);
  }
  printf("\n");
#endif

  for (int b = 0; b < cmprts.n_blocks + 1; b++) {
    int cnt = cbndp->d_spine_cnts[10 * cmprts.n_blocks + b];
    // the particles in cell 3 and 19 went out of bounds
    if (b == 3 || b == 19) {
      EXPECT_EQ(cnt, 1) << "where b = " << b;
    } else {
      EXPECT_EQ(cnt, 0) << "where b = " << b;
    }
  }

#if 0
  printf("sum: ");
  for (int b = 0; b < cmprts.n_blocks + 1; b++) {
    int cnt = cbndp->d_spine_sums[10*cmprts.n_blocks + b];
    printf(" %3d", cnt);
  }
  printf("\n");
#endif

  for (int b = 0; b < cmprts.n_blocks + 1; b++) {
    int cnt = cbndp->d_spine_sums[10 * cmprts.n_blocks + b];
    // the particles in cell 3 and 19 went out of bounds
    if (b <= 3) {
      EXPECT_EQ(cnt, 0) << "where b = " << b;
    } else if (b <= 19) {
      EXPECT_EQ(cnt, 1) << "where b = " << b;
    } else {
      EXPECT_EQ(cnt, 2) << "where b = " << b;
    }
  }

  // test find_n_send
  cbndp->n_prts_send = cbndp->find_n_send(cmprts_.get());

  for (int p = 0; p < cmprts.n_patches(); p++) {
    // printf("p %d: n_send %d\n", p, cmprts.bpatch[p].n_send);
    EXPECT_EQ(cbndp->n_sends[p], p < 2 ? 1 : 0);
  }
  EXPECT_EQ(cbndp->n_prts_send, 2);

  // test scan_send_buf_total
#if 1
  cbndp->scan_send_buf_total(cmprts_.get(), cbndp->n_prts_send);

#if 0
  printf("ids: ");
  for (int n = cmprts.n_prts - cmprts.n_prts_send; n < cmprts.n_prts; n++) {
    int id = cmprts.d_id[n];
    printf(" %3d", id);
  }
  printf("\n");
#endif
  EXPECT_EQ(cmprts.n_prts, 4);
  EXPECT_EQ(cbndp->n_prts_send, 2);
  EXPECT_EQ(cmprts.by_block_.d_id[2], 1);
  EXPECT_EQ(cmprts.by_block_.d_id[3], 3);

#else
  cbndp->scan_send_buf_total_gold(cmprts.get(), cbndp->n_prts_send);
  // the intermediate scan_send_buf_total_gold result
  // can be tested here, but the non-gold version works differently
  // and has different intermediate results
#if 0
  printf("sums: ");
  for (int n = 0; n < cmprts.n_prts; n++) {
    int sum = cmprts.d_sums[n];
    printf(" %3d", sum);
  }
  printf("\n");
#endif

  // where in the send region at the tail the OOB particles should go
  EXPECT_EQ(cbndp->d_sums[1], 0);
  EXPECT_EQ(cbndp->d_sums[3], 1);
#endif

  // particles 1, 3, which need to be exchanged, should now be at the
  // end of the regular array
  EXPECT_EQ(cmprts.storage.load(cmprts.n_prts).kind, 1);
  EXPECT_EQ(cmprts.storage.load(cmprts.n_prts + 1).kind, 3);

  // test copy_from_dev_and_convert
  cbndp->copy_from_dev_and_convert(cmprts_.get(), cbndp->n_prts_send);

#if 0
  for (int p = 0; p < cmprts.n_patches; p++) {
    printf("from_dev: p %d\n", p);
    for (auto& prt : cmprts.bpatch[p].buf) {
      printf("  prt xyz %g %g %g kind %d\n", prt.xi, prt.yi, prt.zi, prt.kind_);
    }
  }
#endif

  EXPECT_EQ(cbndp->bufs[0].size(), 1);
  EXPECT_EQ(cbndp->bufs[1].size(), 1);
  EXPECT_EQ(cbndp->bufs[0][0].kind, 1);
  EXPECT_EQ(cbndp->bufs[1][0].kind, 3);
}

// ----------------------------------------------------------------------
// BndPost
//
// tests cuda_bndp::post()

TEST_F(CudaMparticlesBndTest, BndPost)
{
  auto& cmprts = *cmprts_;

  // BndPost expects the work done by bnd_prep()
  cbndp->prep(cmprts_.get());

  // particles 0 and 2 remain in their patch,
  // particles 1 and 3 leave their patch and need special handling
  EXPECT_EQ(cbndp->bufs[0].size(), 1);
  EXPECT_EQ(cbndp->bufs[1].size(), 1);
  EXPECT_EQ(cbndp->bufs[0][0].kind, 1);
  EXPECT_EQ(cbndp->bufs[1][0].kind, 3);

  // Mock what the actual boundary exchange does, ie., move
  // particles to their new patch and adjust the relative position.
  // This assumes periodic b.c.
  auto prt1 = cbndp->bufs[0][0];
  auto prt3 = cbndp->bufs[1][0];
  prt1.x()[1] -= 40.;
  prt3.x()[1] -= 40.;
  cbndp->bufs[0][0] = prt3;
  cbndp->bufs[1][0] = prt1;

  cbndp->post(cmprts_.get());

  // bnd_post doesn't do the actual final reordering
  EXPECT_TRUE(cmprts.need_reorder);
  cmprts.reorder();
  EXPECT_TRUE(cmprts.check_ordered());

#if 0
  cmprts.dump();
#endif
}

// ----------------------------------------------------------------------
// BndPostDetail
//
// tests the pieces that go into cuda_bndp::post()

TEST_F(CudaMparticlesBndTest, BndPostDetail)
{
  auto& cmprts = *cmprts_;

  // BndPost expects the work done by bnd_prep()
  cbndp->prep(cmprts_.get());

  // particles 0 and 2 remain in their patch,
  // particles 1 and 3 leave their patch and need special handling
  EXPECT_EQ(cbndp->bufs[0].size(), 1);
  EXPECT_EQ(cbndp->bufs[1].size(), 1);
  EXPECT_EQ(cbndp->bufs[0][0].kind, 1);
  EXPECT_EQ(cbndp->bufs[1][0].kind, 3);

  // Mock what the actual boundary exchange does, ie., move
  // particles to their new patch and adjust the relative position.
  // This assumes periodic b.c.
  auto prt1 = cbndp->bufs[0][0];
  auto prt3 = cbndp->bufs[1][0];
  prt1.x()[1] -= 160.;
  prt3.x()[1] -= 160.;
  cbndp->bufs[0][0] = prt3;
  cbndp->bufs[1][0] = prt1;

  // === test convert_and_copy_to_dev()
  uint n_prts_recv = cbndp->convert_and_copy_to_dev(cmprts_.get());
  cmprts.n_prts += n_prts_recv;

  // n_recv should be set for each patch, and its total
  EXPECT_EQ(cbndp->n_recvs[0], 1);
  EXPECT_EQ(cbndp->n_recvs[1], 1);
  EXPECT_EQ(n_prts_recv, 2);

  // the received particle have been added to the previous total
  EXPECT_EQ(cmprts.n_prts, 6);

  // and the particle have been appended after the old end of the particle list
  int n_prts_old = cmprts.n_prts - n_prts_recv;
  EXPECT_EQ(cmprts.storage.load(n_prts_old).kind, 3);
  EXPECT_EQ(cmprts.storage.load(n_prts_old + 1).kind, 1);

  // block indices have been calculated
  auto& d_bidx = cmprts.by_block_.d_idx;
  EXPECT_EQ(d_bidx[n_prts_old], 0);      // 0th block in 0th patch
  EXPECT_EQ(d_bidx[n_prts_old + 1], 16); // 0th block in 1st patch

  // received particles per block have been counted
  for (int b = 0; b < cmprts.n_blocks; b++) {
    if (b == 0 || b == 16) {
      EXPECT_EQ(cbndp->d_spine_cnts[10 * cmprts.n_blocks + b], 1);
    } else {
      EXPECT_EQ(cbndp->d_spine_cnts[10 * cmprts.n_blocks + b], 0);
    }
  }

  // both particles are the 0th (and only) particle added to their respective
  // block
  EXPECT_EQ(cbndp->d_bnd_off[0], 0);
  EXPECT_EQ(cbndp->d_bnd_off[1], 0);

  // === test sort
  auto n_prts_by_patch = cmprts.sizeByPatch();
  EXPECT_EQ(n_prts_by_patch, std::vector<uint>({2, 2, 0, 0}));

  cbndp->sort_pairs_device(cmprts_.get(), n_prts_recv);
  cmprts.n_prts -= cbndp->n_prts_send;

  EXPECT_EQ(cmprts.n_prts, 4);
  auto& d_id = cmprts.by_block_.d_id;
  EXPECT_EQ(d_id[0], 4);
  EXPECT_EQ(d_id[1], 0);
  EXPECT_EQ(d_id[2], 5);
  EXPECT_EQ(d_id[3], 2);

  cbndp->update_offsets(cmprts_.get());
  auto& d_off = cmprts.by_block_.d_off;
  for (int b = 0; b <= cmprts.n_blocks; b++) {
    // if (b < cmprts.n_blocks) printf("b %d: off [%d:%d[\n", b, int(d_off[b]),
    // int(d_off[b+1]));
    if (b < 1) {
      EXPECT_EQ(d_off[b], 0) << "where b = " << b;
    } else if (b < 2) {
      EXPECT_EQ(d_off[b], 1) << "where b = " << b;
    } else if (b < 17) {
      EXPECT_EQ(d_off[b], 2) << "where b = " << b;
    } else if (b < 18) {
      EXPECT_EQ(d_off[b], 3) << "where b = " << b;
    } else {
      EXPECT_EQ(d_off[b], 4) << "where b = " << b;
    }
  }

  cmprts.need_reorder = true;

  // bnd_post doesn't do the actually final reordering, but
  // let's do it here for a final check
  cmprts.reorder();
  EXPECT_TRUE(cmprts.check_ordered());

#if 0
  cmprts.dump();
#endif
}

// ======================================================================
// main

int main(int argc, char** argv)
{
  MPI_Init(&argc, &argv);

  ::testing::InitGoogleTest(&argc, argv);
  int rc = RUN_ALL_TESTS();

  MPI_Finalize();
  return rc;
}
