#include "hip/hip_runtime.h"

#include "cuda_iface.h"
#include "cuda_iface_bnd.h"
#include "cuda_mfields.h"
#include "cuda_bits.h"

#include "psc.h"

#define BLOCKSIZE_X 1
#define BLOCKSIZE_Y 4
#define BLOCKSIZE_Z 4

#define SW (2) // FIXME

// OPT lots of optimization opportunity in the single-proc/patch ones,
// but they may not be that important for float production

__global__ static void fill_ghosts_periodic_yz(DFields d_flds, int mb, int me)
{
  int iy = blockIdx.x * blockDim.x + threadIdx.x;
  int iz = blockIdx.y * blockDim.y + threadIdx.y;

  if (!(iy < d_flds.im(1) && iz < d_flds.im(2)))
    return;

  bool inside = true;
  int jy = iy, jz = iz;
  if (jy < SW) {
    jy += d_flds.im(1) - 2 * SW;
    inside = false;
  }
  if (jy >= d_flds.im(1) - SW) {
    jy -= d_flds.im(1) - 2 * SW;
    inside = false;
  }
  if (jz < SW) {
    jz += d_flds.im(2) - 2 * SW;
    inside = false;
  }
  if (jz >= d_flds.im(2) - SW) {
    jz -= d_flds.im(2) - 2 * SW;
    inside = false;
  }

  if (inside)
    return;

  for (int m = mb; m < me; m++) {
    d_flds(m, 0, iy - SW, iz - SW) = d_flds(m, 0, jy - SW, jz - SW);
  }
}

void cuda_fill_ghosts_periodic_yz(struct cuda_mfields* cmflds, int p, int mb,
                                  int me)
{
  assert(cmflds->ib(1) == -SW);
  assert(cmflds->ib(2) == -SW);

  dim3 dimBlock(BLOCKSIZE_Y, BLOCKSIZE_Z);
  dim3 dimGrid((cmflds->im(1) + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y,
               (cmflds->im(2) + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z);
  fill_ghosts_periodic_yz<<<dimGrid, dimBlock>>>((*cmflds)[p], mb, me);
  cuda_sync_if_enabled();
}

__global__ static void fill_ghosts_periodic_z(DFields d_flds, int mb, int me)
{
  int iy = blockIdx.x * blockDim.x + threadIdx.x;
  int iz = blockIdx.y * blockDim.y + threadIdx.y;

  if (!(iy < d_flds.im(1) && iz < d_flds.im(2)))
    return;

  bool inside = true;
  int jy = iy, jz = iz;
  if (jz < SW) {
    jz += d_flds.im(2) - 2 * SW;
    inside = false;
  }
  if (jz >= d_flds.im(2) - SW) {
    jz -= d_flds.im(2) - 2 * SW;
    inside = false;
  }

  if (inside)
    return;

  for (int m = mb; m < me; m++) {
    d_flds(m, 0, iy - SW, iz - SW) = d_flds(m, 0, jy - SW, jz - SW);
  }
}

void cuda_fill_ghosts_periodic_z(struct cuda_mfields* cmflds, int p, int mb,
                                 int me)
{
  dim3 dimBlock(BLOCKSIZE_Y, BLOCKSIZE_Z);
  dim3 dimGrid((cmflds->im(1) + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y,
               (cmflds->im(2) + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z);
  fill_ghosts_periodic_z<<<dimGrid, dimBlock>>>((*cmflds)[p], mb, me);
  cuda_sync_if_enabled();
}

__global__ static void add_ghosts_periodic_yz(DFields d_flds, int mb, int me)
{
  int iy = blockIdx.x * blockDim.x + threadIdx.x;
  int iz = blockIdx.y * blockDim.y + threadIdx.y;

  if (!(iy < d_flds.im(1) - 2 * SW && iz < d_flds.im(2) - 2 * SW))
    return;

  if (iy < SW) {
    int jy = iy + (d_flds.im(1) - 2 * SW);
    int jz = iz;
    for (int m = mb; m < me; m++) {
      d_flds(m, 0, iy, iz) += d_flds(m, 0, jy, jz);
    }
    if (iz < SW) {
      jz = iz + (d_flds.im(2) - 2 * SW);
      for (int m = mb; m < me; m++) {
        d_flds(m, 0, iy, iz) += d_flds(m, 0, jy, jz);
      }
    }
    if (iz >= d_flds.im(2) - 3 * SW) {
      jz = iz - (d_flds.im(2) - 2 * SW);
      for (int m = mb; m < me; m++) {
        d_flds(m, 0, iy, iz) += d_flds(m, 0, jy, jz);
      }
    }
  }
  if (iy >= d_flds.im(1) - 3 * SW) {
    int jy = iy - (d_flds.im(1) - 2 * SW);
    int jz = iz;
    for (int m = mb; m < me; m++) {
      d_flds(m, 0, iy, iz) += d_flds(m, 0, jy, jz);
    }
    if (iz < SW) {
      jz = iz + (d_flds.im(2) - 2 * SW);
      for (int m = mb; m < me; m++) {
        d_flds(m, 0, iy, iz) += d_flds(m, 0, jy, jz);
      }
    }
    if (iz >= d_flds.im(2) - 3 * SW) {
      jz = iz - (d_flds.im(2) - 2 * SW);
      for (int m = mb; m < me; m++) {
        d_flds(m, 0, iy, iz) += d_flds(m, 0, jy, jz);
      }
    }
  }
  if (iz < SW) {
    int jy = iy, jz = iz + (d_flds.im(2) - 2 * SW);
    for (int m = mb; m < me; m++) {
      d_flds(m, 0, iy, iz) += d_flds(m, 0, jy, jz);
    }
  }
  if (iz >= d_flds.im(2) - 3 * SW) {
    int jy = iy, jz = iz - (d_flds.im(2) - 2 * SW);
    for (int m = mb; m < me; m++) {
      d_flds(m, 0, iy, iz) += d_flds(m, 0, jy, jz);
    }
  }
}

void cuda_add_ghosts_periodic_yz(struct cuda_mfields* cmflds, int p, int mb,
                                 int me)
{
  Int3 ldims = cmflds->grid().ldims;
  dim3 dimBlock(BLOCKSIZE_Y, BLOCKSIZE_Z);
  dim3 dimGrid((ldims[1] + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y,
               (ldims[2] + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z);

  add_ghosts_periodic_yz<<<dimGrid, dimBlock>>>((*cmflds)[p], mb, me);
  cuda_sync_if_enabled();
}

__global__ static void add_ghosts_periodic_z(DFields d_flds, int mb, int me)
{
  int iy = blockIdx.x * blockDim.x + threadIdx.x;
  int iz = blockIdx.y * blockDim.y + threadIdx.y;

  if (!(iy < d_flds.im(1) - 2 * SW && iz < d_flds.im(2) - 2 * SW))
    return;

  if (iz < SW) {
    int jy = iy, jz = iz + (d_flds.im(2) - 2 * SW);
    for (int m = mb; m < me; m++) {
      d_flds(m, 0, iy, iz) += d_flds(m, 0, jy, jz);
    }
  }
  if (iz >= d_flds.im(2) - 3 * SW) {
    int jy = iy, jz = iz - (d_flds.im(2) - 2 * SW);
    for (int m = mb; m < me; m++) {
      d_flds(m, 0, iy, iz) += d_flds(m, 0, jy, jz);
    }
  }
}

void cuda_add_ghosts_periodic_z(struct cuda_mfields* cmflds, int p, int mb,
                                int me)
{
  Int3 ldims = cmflds->grid().ldims;
  dim3 dimBlock(BLOCKSIZE_Y, BLOCKSIZE_Z);
  dim3 dimGrid((ldims[1] + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y,
               (ldims[2] + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z);
  add_ghosts_periodic_z<<<dimGrid, dimBlock>>>((*cmflds)[p], mb, me);
  cuda_sync_if_enabled();
}

template <bool lo, bool hi>
__global__ static void conducting_wall_H_y(DFields d_flds)
{
  int iz = blockIdx.x * blockDim.x + threadIdx.x - SW;

  if (iz >= d_flds.im(2) - SW)
    return;

  int my = d_flds.im(1) - 2 * SW;

  if (lo) {
    d_flds(HY, 0, -1, iz) = d_flds(HY, 0, 1, iz);
    d_flds(HX, 0, -1, iz) = -d_flds(HX, 0, 0, iz);
    d_flds(HZ, 0, -1, iz) = -d_flds(HZ, 0, 0, iz);
  }

  if (hi) {
    d_flds(HY, 0, my + 1, iz) = d_flds(HY, 0, my - 1, iz);
    d_flds(HX, 0, my, iz) = -d_flds(HX, 0, my - 1, iz);
    d_flds(HZ, 0, my, iz) = -d_flds(HZ, 0, my - 1, iz);
  }
}

template <bool lo, bool hi>
__global__ static void conducting_wall_E_y(DFields d_flds)
{
  int iz = blockIdx.x * blockDim.x + threadIdx.x - SW;

  if (iz >= d_flds.im(2) - SW)
    return;

  int my = d_flds.im(1) - 2 * SW;

  if (lo) {
    d_flds(EX, 0, 0, iz) = 0.;
    d_flds(EX, 0, -1, iz) = d_flds(EX, 0, 1, iz);
    d_flds(EY, 0, -1, iz) = -d_flds(EY, 0, 0, iz);
    d_flds(EZ, 0, 0, iz) = 0.;
    d_flds(EZ, 0, -1, iz) = d_flds(EZ, 0, 1, iz);
  }

  if (hi) {
    d_flds(EX, 0, my, iz) = 0.;
    d_flds(EX, 0, my + 1, iz) = d_flds(EX, 0, my - 1, iz);
    d_flds(EY, 0, my, iz) = -d_flds(EY, 0, my - 1, iz);
    d_flds(EZ, 0, my, iz) = 0.;
    d_flds(EZ, 0, my + 1, iz) = d_flds(EZ, 0, my - 1, iz);
  }
}

template <bool lo, bool hi>
__global__ static void conducting_wall_J_y(DFields d_flds)
{
  int iz = blockIdx.x * blockDim.x + threadIdx.x - SW;

  if (iz >= d_flds.im(2) - SW)
    return;

  int my = d_flds.im(1) - 2 * SW;

  if (lo) {
    d_flds(JYI, 0, 0, iz) -= d_flds(JYI, 0, -1, iz);
    d_flds(JYI, 0, -1, iz) = 0.;
    d_flds(JXI, 0, 1, iz) += d_flds(JXI, 0, -1, iz);
    d_flds(JXI, 0, -1, iz) = 0.;
    d_flds(JZI, 0, 1, iz) += d_flds(JZI, 0, -1, iz);
    d_flds(JZI, 0, -1, iz) = 0.;
  }

  if (hi) {
    d_flds(JYI, 0, my - 1, iz) -= d_flds(JYI, 0, my, iz);
    d_flds(JYI, 0, my, iz) = 0.;
    d_flds(JXI, 0, my - 1, iz) += d_flds(JXI, 0, my + 1, iz);
    d_flds(JXI, 0, my + 1, iz) = 0.;
    d_flds(JZI, 0, my - 1, iz) += d_flds(JZI, 0, my + 1, iz);
    d_flds(JZI, 0, my + 1, iz) = 0.;
  }
}

template <bool lo, bool hi>
static void cuda_conducting_wall_H_y(struct cuda_mfields* cmflds, int p)
{
  int dimGrid = (cmflds->im(2) + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z;
  conducting_wall_H_y<lo, hi><<<dimGrid, BLOCKSIZE_Z>>>((*cmflds)[p]);
  cuda_sync_if_enabled();
}

template <bool lo, bool hi>
static void cuda_conducting_wall_E_y(struct cuda_mfields* cmflds, int p)
{
  int dimGrid = (cmflds->im(2) + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z;
  conducting_wall_E_y<lo, hi><<<dimGrid, BLOCKSIZE_Z>>>((*cmflds)[p]);
  cuda_sync_if_enabled();
}

template <bool lo, bool hi>
static void cuda_conducting_wall_J_y(struct cuda_mfields* cmflds, int p)
{
  int dimGrid = (cmflds->im(2) + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z;
  conducting_wall_J_y<lo, hi><<<dimGrid, BLOCKSIZE_Z>>>((*cmflds)[p]);
  cuda_sync_if_enabled();
}

void cuda_conducting_wall_H_lo_y(struct cuda_mfields* cmflds, int p)
{
  cuda_conducting_wall_H_y<true, false>(cmflds, p);
}

void cuda_conducting_wall_H_hi_y(struct cuda_mfields* cmflds, int p)
{
  cuda_conducting_wall_H_y<false, true>(cmflds, p);
}

void cuda_conducting_wall_H_lo_hi_y(struct cuda_mfields* cmflds, int p)
{
  cuda_conducting_wall_H_y<true, true>(cmflds, p);
}

void cuda_conducting_wall_E_lo_y(struct cuda_mfields* cmflds, int p)
{
  cuda_conducting_wall_E_y<true, false>(cmflds, p);
}

void cuda_conducting_wall_E_hi_y(struct cuda_mfields* cmflds, int p)
{
  cuda_conducting_wall_E_y<false, true>(cmflds, p);
}

void cuda_conducting_wall_E_lo_hi_y(struct cuda_mfields* cmflds, int p)
{
  cuda_conducting_wall_E_y<true, true>(cmflds, p);
}

void cuda_conducting_wall_J_lo_y(struct cuda_mfields* cmflds, int p)
{
  cuda_conducting_wall_J_y<true, false>(cmflds, p);
}

void cuda_conducting_wall_J_hi_y(struct cuda_mfields* cmflds, int p)
{
  cuda_conducting_wall_J_y<false, true>(cmflds, p);
}

void cuda_conducting_wall_J_lo_hi_y(struct cuda_mfields* cmflds, int p)
{
  cuda_conducting_wall_J_y<true, true>(cmflds, p);
}
