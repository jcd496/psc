
#include "cuda_iface.h"
#include "cuda_mfields.h"
#include "cuda_bits.h"

#include "psc_fields_cuda.h"

#undef dprintf
#if 0
#define dprintf(...) mprintf(__VA_ARGS__)
#else
#define dprintf(...)                                                           \
  do {                                                                         \
  } while (0)
#endif

MfieldsCuda::MfieldsCuda(const Grid_t& grid, int n_fields, Int3 ibn)
  : MfieldsBase{grid, n_fields, ibn}, grid_{&grid}
{
  dprintf("CMFLDS: ctor\n");
  cmflds_ = new cuda_mfields(grid, n_fields, ibn);
}

MfieldsCuda::~MfieldsCuda()
{
  dprintf("CMFLDS: dtor\n");
  delete cmflds_;
}

int MfieldsCuda::n_comps() const
{
  return cmflds_->n_comps();
}

int MfieldsCuda::n_patches() const
{
  return cmflds_->n_patches();
}

void MfieldsCuda::reset(const Grid_t& new_grid)
{
  dprintf("CMFLDS: reset\n");
  MfieldsBase::reset(new_grid);
  Int3 ibn = -cmflds()->ib();
  int n_comps = cmflds()->n_comps();
  delete cmflds_;
  cmflds_ = new cuda_mfields(new_grid, n_comps, ibn);
  grid_ = &new_grid;
}

void MfieldsCuda::copy_comp(int ym, MfieldsCuda& mflds_x, int xm)
{
  dprintf("CMFLDS: copy_comp_\n");
  cmflds()->copy_comp(ym, mflds_x.cmflds(), xm);
}

void MfieldsCuda::axpy_comp(int ym, float a, MfieldsCuda& mflds_x, int xm)
{
  dprintf("CMFLDS: axpy_comp\n");
  cmflds()->axpy_comp(ym, a, mflds_x.cmflds(), xm);
}

void MfieldsCuda::zero_comp(int m)
{
  dprintf("CMFLDS: zero_comp\n");
  assert(!grid_->isInvar(1));
  assert(!grid_->isInvar(2));
  if (grid_->isInvar(0)) {
    cmflds()->zero_comp(m, dim_yz{});
  } else {
    cmflds()->zero_comp(m, dim_xyz{});
  }
}

void MfieldsCuda::zero()
{
  dprintf("CMFLDS: zero\n");
  for (int m = 0; m < cmflds()->n_comps(); m++) {
    zero_comp(m);
  }
}

int MfieldsCuda::index(int m, int i, int j, int k, int p) const
{
  return cmflds_->index(m, i, j, k, p);
}

MfieldsCuda::Patch::Patch(MfieldsCuda& mflds, int p) : mflds_(mflds), p_(p) {}

MfieldsCuda::Accessor MfieldsCuda::Patch::operator()(int m, int i, int j, int k)
{
  return {mflds_, mflds_.index(m, i, j, k, p_)};
}

MfieldsCuda::Accessor::Accessor(MfieldsCuda& mflds, int idx)
  : mflds_(mflds), idx_(idx)
{}

MfieldsCuda::Accessor::operator real_t() const
{
  return mflds_.cmflds_->get_value(idx_);
}

MfieldsCuda::real_t MfieldsCuda::Accessor::operator=(real_t val)
{
  mflds_.cmflds_->set_value(idx_, val);
  return val;
}

MfieldsCuda::real_t MfieldsCuda::Accessor::operator+=(real_t val)
{
  val += mflds_.cmflds_->get_value(idx_);
  mflds_.cmflds_->set_value(idx_, val);
  return val;
}

HMFields hostMirror(const MfieldsCuda& mflds)
{
  return hostMirror(*mflds.cmflds());
}

void copy(const MfieldsCuda& mflds, HMFields& hmflds)
{
  copy(*mflds.cmflds(), hmflds);
}

void copy(const HMFields& hmflds, MfieldsCuda& mflds)
{
  copy(hmflds, *mflds.cmflds());
}
