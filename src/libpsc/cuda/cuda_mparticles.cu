#include "hip/hip_runtime.h"

#include "cuda_mparticles.cuh"
#include "cuda_bits.h"

#include "psc_bits.h"
#include "bs.hxx"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include "cuda_base.cuh"

#include <cstdio>
#include <cassert>

// ----------------------------------------------------------------------
// ctor

template <typename BS>
cuda_mparticles<BS>::cuda_mparticles(const Grid_t& grid)
  : cuda_mparticles_base<BS>(grid)
{
  cuda_base_init();

  xb_by_patch.resize(this->n_patches());
  for (int p = 0; p < this->n_patches(); p++) {
    xb_by_patch[p] = Real3(grid.patches[p].xb);
  }
}

// ----------------------------------------------------------------------
// resize
//
// the goal here is to have d_xi4, d_pxi4, d_bidx and d_id always
// have the same size.

template <typename BS>
void cuda_mparticles<BS>::resize(uint n_prts)
{
  cuda_mparticles_base<BS>::resize(n_prts);
  this->by_block_.d_idx.resize(n_prts);
  this->by_block_.d_id.resize(n_prts);
}

// ----------------------------------------------------------------------
// dump_by_patch

template <typename BS>
void cuda_mparticles<BS>::dump_by_patch(uint* n_prts_by_patch)
{
  printf("cuda_mparticles_dump_by_patch: n_prts = %d\n", this->n_prts);
  uint off = 0;
  for (int p = 0; p < this->n_patches(); p++) {
    float* xb = &xb_by_patch[p][0];
    for (int n = 0; n < n_prts_by_patch[p]; n++) {
      auto prt = this->storage.load(n + off);
      uint bidx = this->by_block_.d_idx[n + off],
           id = this->by_block_.d_id[n + off];
      printf("cuda_mparticles_dump_by_patch: [%d/%d] %g %g %g // %d // %g %g "
             "%g // %g b_idx %d id %d\n",
             p, n, prt.x[0] + xb[0], prt.x[1] + xb[1], prt.x[2] + xb[2],
             prt.kind, prt.u[0], prt.u[1], prt.u[2], prt.qni_wni, bidx, id);
    }
    off += n_prts_by_patch[p];
  }
}

// ----------------------------------------------------------------------
// dump

template <typename BS>
void cuda_mparticles<BS>::dump(const std::string& filename) const
{
  FILE* file = fopen(filename.c_str(), "w");
  assert(file);

  fprintf(file, "cuda_mparticles_dump: n_prts = %d\n", this->n_prts);
  uint off = 0;
  auto& d_off = this->by_block_.d_off;
  for (int b = 0; b < this->n_blocks; b++) {
    uint off_b = d_off[b], off_e = d_off[b + 1];
    int p = b / this->n_blocks_per_patch;
    fprintf(file, "cuda_mparticles_dump: block %d: %d -> %d (patch %d)\n", b,
            off_b, off_e, p);
    assert(d_off[b] == off);
    for (int n = d_off[b]; n < d_off[b + 1]; n++) {
      auto prt = this->storage.load(n + off);
      uint bidx = this->by_block_.d_idx[n], id = this->by_block_.d_id[n];
      fprintf(file,
              "mparticles_dump: [%d] %g %g %g // %d // %g %g %g // %g || bidx "
              "%d id %d %s\n",
              n, prt.x[0], prt.x[1], prt.x[2], prt.kind, prt.u[0], prt.u[1],
              prt.u[2], prt.qni_wni, bidx, id,
              b == bidx ? "" : "BIDX MISMATCH!");
    }
    off += off_e - off_b;
  }
  fclose(file);
}

// ----------------------------------------------------------------------
// swap_alt

template <typename BS>
void cuda_mparticles<BS>::swap_alt()
{
  this->storage.xi4.swap(alt_storage.xi4);
  // thrust::swap(this->storage.xi4, alt_storage.xi4);
  this->storage.pxi4.swap(alt_storage.pxi4);
  // thrust::swap(this->storage.pxi4, alt_storage.pxi4);
}

#define THREADS_PER_BLOCK 256

// ----------------------------------------------------------------------
// k_reorder_and_offsets

template <typename BS>
__global__ static void k_reorder_and_offsets(DMparticlesCuda<BS> dmprts,
                                             int nr_prts, const uint* d_bidx,
                                             const uint* d_ids, uint* d_off,
                                             int last_block)
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;

  for (; i <= nr_prts; i += blockDim.x * gridDim.x) {
    int block, prev_block;
    if (i < nr_prts) {
      dmprts.storage.xi4[i] = dmprts.alt_storage.xi4[d_ids[i]];
      dmprts.storage.pxi4[i] = dmprts.alt_storage.pxi4[d_ids[i]];

      block = d_bidx[i];
    } else { // needed if there is no particle in the last block
      block = last_block;
    }

    // OPT: d_bidx[i-1] could use shmem
    // create offsets per block into particle array
    prev_block = -1;
    if (i > 0) {
      prev_block = d_bidx[i - 1];
    }
    for (int b = prev_block + 1; b <= block; b++) {
      d_off[b] = i;
    }
  }
}

// ----------------------------------------------------------------------
// reorder_and_offsets

template <typename BS>
void cuda_mparticles<BS>::reorder_and_offsets(
  const psc::device_vector<uint>& d_idx, const psc::device_vector<uint>& d_id,
  psc::device_vector<uint>& d_off)
{
  if (this->n_patches() == 0) {
    return;
  }

  swap_alt();
  resize(this->n_prts);

  int n_blocks = (this->n_prts + 1 + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  if (n_blocks > 32768)
    n_blocks = 32768;
  dim3 dimGrid(n_blocks);
  dim3 dimBlock(THREADS_PER_BLOCK);

  k_reorder_and_offsets<BS><<<dimGrid, dimBlock>>>(
    *this, this->n_prts, d_idx.data().get(), d_id.data().get(),
    d_off.data().get(), this->n_blocks);
  cuda_sync_if_enabled();

  need_reorder = false;
}

// ----------------------------------------------------------------------
// k_reorder

template <typename BS>
__global__ static void k_reorder(DMparticlesCuda<BS> dmprts, int n_prts,
                                 const uint* d_ids)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;

  if (i < n_prts) {
    int j = d_ids[i];
    dmprts.storage.xi4[i] = dmprts.alt_storage.xi4[j];
    dmprts.storage.pxi4[i] = dmprts.alt_storage.pxi4[j];
  }
}

// ----------------------------------------------------------------------
// reorder

template <typename BS>
void cuda_mparticles<BS>::reorder()
{
  if (!need_reorder) {
    return;
  }

  reorder(this->by_block_.d_id);
  need_reorder = false;
}

// ----------------------------------------------------------------------
// reorder

template <typename BS>
void cuda_mparticles<BS>::reorder(const psc::device_vector<uint>& d_id)
{
  if (this->n_prts == 0) {
    return;
  }

  swap_alt();
  resize(this->n_prts);

  dim3 dimGrid((this->n_prts + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);

  k_reorder<BS>
    <<<dimGrid, THREADS_PER_BLOCK>>>(*this, this->n_prts, d_id.data().get());
  cuda_sync_if_enabled();
}

// ----------------------------------------------------------------------
// setup_internals

template <typename BS>
void cuda_mparticles<BS>::setup_internals()
{
  // pre-condition: particles sorted by patch, d_off being used to
  // describe patch boundaries

  // assert(check_in_patch_unordered_slow());

  this->by_block_.find_indices_ids(*this);

  // assert(check_bidx_id_unordered_slow());

  this->by_block_.stable_sort();

  this->by_block_.reorder_and_offsets(*this);

  // post-condition:
  // - particles now sorted by block
  // - d_off describes block boundaries
  // - UNUSED: d_bidx has each particle's block index

  // assert(check_ordered());
}

// ----------------------------------------------------------------------
// size

template <typename BS>
uint cuda_mparticles<BS>::size()
{
  return this->n_prts;
}

// ----------------------------------------------------------------------
// inject_initial
//
// adds particles initially, ie., into an empty cmprts
// does not complete setting correct internal state
// (setup_internal() needs to be called next)

template <typename BS>
void cuda_mparticles<BS>::inject_initial(
  const std::vector<Particle>& buf, const std::vector<uint>& n_prts_by_patch)
{
  thrust::host_vector<uint> h_off(this->by_block_.d_off);

  assert(this->storage.xi4.size() == 0);
  assert(this->n_prts == 0);

  uint buf_n = 0;
  for (int p = 0; p < this->n_patches(); p++) {
    assert(h_off[p * this->n_blocks_per_patch] == 0);
    assert(h_off[(p + 1) * this->n_blocks_per_patch] == 0);
    buf_n += n_prts_by_patch[p];
  }

  resize(buf_n);

  HMparticlesCudaStorage h_storage{buf_n};

  auto it = buf.begin();
  uint off = 0;
  for (int p = 0; p < this->n_patches(); p++) {
    auto n_prts = n_prts_by_patch[p];
    h_off[p * this->n_blocks_per_patch] = off;
    h_off[(p + 1) * this->n_blocks_per_patch] = off + n_prts;

    for (int n = 0; n < n_prts; n++) {
      auto prt = *it++;
      this->checkInPatchMod(prt.x);
      h_storage.store(prt, off + n);
    }

    off += n_prts;
  }
  this->n_prts = off;

  thrust::copy(h_storage.xi4.begin(), h_storage.xi4.end(),
               this->storage.xi4.begin());
  thrust::copy(h_storage.pxi4.begin(), h_storage.pxi4.end(),
               this->storage.pxi4.begin());
  thrust::copy(h_off.begin(), h_off.end(), this->by_block_.d_off.begin());
}

// ----------------------------------------------------------------------
// inject

template <typename BS>
void cuda_mparticles<BS>::inject(const std::vector<Particle>& buf,
                                 const std::vector<uint>& buf_n_by_patch)
{
  if (this->n_prts == 0) {
    // if there are no particles yet, we basically just initialize from the
    // buffer
    inject_initial(buf, buf_n_by_patch);
    setup_internals();
    return;
  }

  using Double3 = Vec3<double>;

  uint buf_n = 0;
  for (int p = 0; p < this->n_patches(); p++) {
    buf_n += buf_n_by_patch[p];
    //    printf("p %d buf_n_by_patch %d\n", p, buf_n_by_patch[p]);
  }
  //  printf("buf_n %d\n", buf_n);

  HMparticlesCudaStorage h_storage(buf_n);
  thrust::host_vector<uint> h_bidx(buf_n);
  // thrust::host_vector<uint> h_id(buf_n);

  uint off = 0;
  for (int p = 0; p < this->n_patches(); p++) {
    for (int n = 0; n < buf_n_by_patch[p]; n++) {
      auto prt = buf[off + n];
      h_storage.store(prt, off + n);
      auto bidx = this->blockIndex(prt, p);
      assert(bidx >= 0 && bidx < this->n_blocks);
      h_bidx[off + n] = bidx;
      ;
      // h_id[off + n] = this->n_prts + off + n;
    }
    off += buf_n_by_patch[p];
  }
  assert(off == buf_n);

  if (need_reorder) {
    reorder();
  }

  // assert(check_in_patch_unordered_slow());

  this->by_block_.find_indices_ids(*this);
  // assert(check_bidx_id_unordered_slow());

  resize(this->n_prts + buf_n);

  thrust::copy(h_storage.xi4.begin(), h_storage.xi4.end(),
               this->storage.xi4.begin() + this->n_prts);
  thrust::copy(h_storage.pxi4.begin(), h_storage.pxi4.end(),
               this->storage.pxi4.begin() + this->n_prts);
  thrust::copy(h_bidx.begin(), h_bidx.end(),
               this->by_block_.d_idx.begin() + this->n_prts);
  // thrust::copy(h_id.begin(), h_id.end(), d_id + n_prts);
  // FIXME, looks like ids up until n_prts have already been set above
  thrust::sequence(this->by_block_.d_id.data(),
                   this->by_block_.d_id.data() + this->n_prts + buf_n);

  // for (int i = -5; i <= 5; i++) {
  //   //    float4 xi4 = d_xi4[cmprts->n_prts + i];
  //   uint bidx = d_bidx[cmprts->n_prts + i];
  //   uint id = d_id[cmprts->n_prts + i];
  //   printf("i %d bidx %d %d\n", i, bidx, id);
  // }

  // assert(check_ordered());

  this->n_prts += buf_n;

  this->by_block_.stable_sort();

  this->by_block_.reorder_and_offsets(*this);

  // assert(check_ordered());
}

// ----------------------------------------------------------------------
// get_particles

template <typename BS>
std::vector<typename cuda_mparticles<BS>::Particle>
cuda_mparticles<BS>::get_particles(int beg, int end)
{
  int n_prts = end - beg;
  std::vector<Particle> prts;
  prts.reserve(n_prts);

  reorder(); // FIXME? by means of this, this function disturbs the state...

  thrust::host_vector<float4> xi4(&this->storage.xi4[beg],
                                  &this->storage.xi4[end]);
  thrust::host_vector<float4> pxi4(&this->storage.pxi4[beg],
                                   &this->storage.pxi4[end]);

  for (int n = 0; n < n_prts; n++) {
    int kind = cuda_float_as_int(xi4[n].w);
    prts.emplace_back(Real3{xi4[n].x, xi4[n].y, xi4[n].z},
                      Real3{pxi4[n].x, pxi4[n].y, pxi4[n].z}, pxi4[n].w, kind,
                      psc::particle::Id(), psc::particle::Tag());

#if 0
    uint b = blockIndex(xi4[n], p);
    assert(b < n_blocks);
#endif
  }

  return prts;
}

// ----------------------------------------------------------------------
// get_particles

template <typename BS>
std::vector<uint> cuda_mparticles<BS>::get_offsets() const
{
  thrust::host_vector<uint> h_off(this->by_block_.d_off);
  std::vector<uint> off(this->n_patches() + 1);
  for (int p = 0; p <= this->n_patches(); p++) {
    off[p] = h_off[p * this->n_blocks_per_patch];
  }
  return off;
}

// ----------------------------------------------------------------------
// get_particles

template <typename BS>
std::vector<typename cuda_mparticles<BS>::Particle>
cuda_mparticles<BS>::get_particles()
{
  return get_particles(0, this->n_prts);
}

// ----------------------------------------------------------------------
// get_particles

template <typename BS>
std::vector<typename cuda_mparticles<BS>::Particle>
cuda_mparticles<BS>::get_particles(int p)
{
  // FIXME, doing the copy here all the time would be nice to avoid
  // making sure we actually have a valid d_off would't hurt, either
  thrust::host_vector<uint> h_off(this->by_block_.d_off);

  uint beg = h_off[p * this->n_blocks_per_patch];
  uint end = h_off[(p + 1) * this->n_blocks_per_patch];

  return get_particles(beg, end);
}

// ----------------------------------------------------------------------
// get_particle

template <typename BS>
typename cuda_mparticles<BS>::Particle cuda_mparticles<BS>::get_particle(int p,
                                                                         int n)
{
  auto off = this->by_block_.d_off[p * this->n_blocks_per_patch];
  auto cprts = get_particles(off + n, off + n + 1);
  return cprts[0];
}

#include "cuda_mparticles_gold.cu"
#include "cuda_mparticles_checks.cu"

template struct cuda_mparticles<BS144>;
template struct cuda_mparticles<BS444>;
