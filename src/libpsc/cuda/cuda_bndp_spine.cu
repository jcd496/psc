
#include <hip/hip_runtime.h>

#ifdef CUDA_BNDP_DIM_YZ_SPECIAL

#include "cuda_bits.h"
#include "cuda_bndp.h"
#include "cuda_mparticles.cuh"

//#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>

#include "b40c/radixsort_reduction_kernel.h"
#include "b40c/radixsort_scanscatter_kernel3.h"

#include <mrc_profile.h>

using namespace b40c_thrust;

typedef uint K;
typedef uint V;

static const int RADIX_BITS = 4;

#define THREADS_PER_BLOCK 256

// ----------------------------------------------------------------------
// spine_reduce

template <typename CudaMparticles>
void cuda_bndp<CudaMparticles, dim_yz>::spine_reduce(CudaMparticles* cmprts)
{
  // OPT?
  thrust::fill(d_spine_cnts.data(),
               d_spine_cnts.data() + 1 + n_blocks * (CUDA_BND_STRIDE + 1), 0);

  const int threads = B40C_RADIXSORT_THREADS;
  Int3 mx = b_mx();
  if (mx[0] == 1 && mx[1] == 2 && mx[2] == 2) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0, NopFunctor<K>, 2, 2>
      <<<n_blocks, threads>>>(d_spine_cnts.data().get(),
                              cmprts->by_block_.d_idx.data().get(),
                              cmprts->by_block_.d_off.data().get(), n_blocks);
  } else if (mx[0] == 1 && mx[1] == 4 && mx[2] == 4) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0, NopFunctor<K>, 4, 4>
      <<<n_blocks, threads>>>(d_spine_cnts.data().get(),
                              cmprts->by_block_.d_idx.data().get(),
                              cmprts->by_block_.d_off.data().get(), n_blocks);
  } else if (mx[0] == 1 && mx[1] == 8 && mx[2] == 8) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0, NopFunctor<K>, 8, 8>
      <<<n_blocks, threads>>>(d_spine_cnts.data().get(),
                              cmprts->by_block_.d_idx.data().get(),
                              cmprts->by_block_.d_off.data().get(), n_blocks);
  } else if (mx[0] == 1 && mx[1] == 16 && mx[2] == 16) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0, NopFunctor<K>, 16, 16>
      <<<n_blocks, threads>>>(d_spine_cnts.data().get(),
                              cmprts->by_block_.d_idx.data().get(),
                              cmprts->by_block_.d_off.data().get(), n_blocks);
  } else if (mx[0] == 1 && mx[1] == 32 && mx[2] == 32) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0, NopFunctor<K>, 32, 32>
      <<<n_blocks, threads>>>(d_spine_cnts.data().get(),
                              cmprts->by_block_.d_idx.data().get(),
                              cmprts->by_block_.d_off.data().get(), n_blocks);
  } else if (mx[0] == 1 && mx[1] == 64 && mx[2] == 64) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0, NopFunctor<K>, 64, 64>
      <<<n_blocks, threads>>>(d_spine_cnts.data().get(),
                              cmprts->by_block_.d_idx.data().get(),
                              cmprts->by_block_.d_off.data().get(), n_blocks);
  } else if (mx[0] == 1 && mx[1] == 128 && mx[2] == 128) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0, NopFunctor<K>, 128, 128>
      <<<n_blocks, threads>>>(d_spine_cnts.data().get(),
                              cmprts->by_block_.d_idx.data().get(),
                              cmprts->by_block_.d_off.data().get(), n_blocks);
  } else {
    printf("no support for b_mx %d x %d x %d!\n", mx[0], mx[1], mx[2]);
    assert(0);
  }
  cuda_sync_if_enabled();

  thrust::exclusive_scan(d_spine_cnts.data() + n_blocks * 10,
                         d_spine_cnts.data() + n_blocks * 10 + n_blocks + 1,
                         d_spine_sums.data() + n_blocks * 10);
}

// ----------------------------------------------------------------------
// cuda_mprts_spine_reduce_gold

template <typename CudaMparticles>
void cuda_bndp<CudaMparticles, dim_yz>::spine_reduce_gold(
  CudaMparticles* cmprts)
{
  thrust::fill(d_spine_cnts.data(),
               d_spine_cnts.data() + 1 + n_blocks * (CUDA_BND_STRIDE + 1), 0);

  thrust::host_vector<uint> h_bidx(cmprts->by_block_.d_idx.data(),
                                   cmprts->by_block_.d_idx.data() +
                                     cmprts->n_prts);
  thrust::host_vector<uint> h_off(cmprts->by_block_.d_off);
  thrust::host_vector<uint> h_spine_cnts(d_spine_cnts.data(),
                                         d_spine_cnts.data() + 1 +
                                           n_blocks * (CUDA_BND_STRIDE + 1));

  Int3 mx = b_mx();
  for (int p = 0; p < n_patches(); p++) {
    for (int b = 0; b < n_blocks_per_patch; b++) {
      uint bid = b + p * n_blocks_per_patch;
      for (int n = h_off[bid]; n < h_off[bid + 1]; n++) {
        uint key = h_bidx[n];
        if (key < 9) {
          int dy = key % 3;
          int dz = key / 3;
          int by = b % mx[1];
          int bz = b / mx[1];
          uint bby = by + 1 - dy;
          uint bbz = bz + 1 - dz;
          uint bb = bbz * mx[1] + bby;
          if (bby < mx[1] && bbz < mx[2]) {
            h_spine_cnts[(bb + p * n_blocks_per_patch) * 10 + key]++;
          } else {
            assert(0);
          }
        } else if (key == CUDA_BND_S_OOB) {
          h_spine_cnts[mx[1] * mx[2] * n_patches() * 10 + bid]++;
        }
      }
    }
  }

  thrust::copy(h_spine_cnts.begin(), h_spine_cnts.end(), d_spine_cnts.begin());
  thrust::exclusive_scan(d_spine_cnts.data() + n_blocks * 10,
                         d_spine_cnts.data() + n_blocks * 10 + n_blocks + 1,
                         d_spine_sums.data() + n_blocks * 10);
}

// ----------------------------------------------------------------------
// k_count_received

__global__ static void k_count_received(int nr_total_blocks,
                                        uint* d_n_recv_by_block,
                                        uint* d_spine_cnts)
{
  int bid = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;

  if (bid < nr_total_blocks) {
    d_spine_cnts[bid * 10 + CUDA_BND_S_NEW] = d_n_recv_by_block[bid];
  }
}

// ----------------------------------------------------------------------
// count_received

template <typename CudaMparticles>
void cuda_bndp<CudaMparticles, dim_yz>::count_received(CudaMparticles* cmprts)
{
  k_count_received<<<n_blocks, THREADS_PER_BLOCK>>>(
    n_blocks, d_spine_cnts.data().get() + 10 * n_blocks,
    d_spine_cnts.data().get());
}

// ----------------------------------------------------------------------
// count_received_gold

template <typename CudaMparticles>
void cuda_bndp<CudaMparticles, dim_yz>::count_received_gold(
  CudaMparticles* cmprts)
{
  thrust::host_vector<uint> h_spine_cnts(1 + n_blocks * (10 + 1));

  thrust::copy(d_spine_cnts.data(),
               d_spine_cnts.data() + 1 + n_blocks * (10 + 1),
               h_spine_cnts.begin());

  for (int bid = 0; bid < n_blocks; bid++) {
    h_spine_cnts[bid * 10 + CUDA_BND_S_NEW] = h_spine_cnts[10 * n_blocks + bid];
  }

  thrust::copy(h_spine_cnts.begin(), h_spine_cnts.end(), d_spine_cnts.begin());
}

#if 0
void cuda_bndp::count_received_v1(CudaMparticles *cmprts)
{
  thrust::device_ptr<uint> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<uint> d_spine_cnts(d_bnd_spine_cnts);

  thrust::host_vector<uint> h_bidx(cmprts->n_prts);
  thrust::host_vector<uint> h_spine_cnts(1 + n_blocks * (10 + 1));

  thrust::copy(d_bidx, d_bidx + cmprts->n_prts, h_bidx.begin());
  thrust::copy(d_spine_cnts, d_spine_cnts + 1 + n_blocks * (10 + 1), h_spine_cnts.begin());
  for (int n = cmprts->n_prts - n_prts_recv; n < cmprts->n_prts; n++) {
    assert(h_bidx[n] < n_blocks);
    h_spine_cnts[h_bidx[n] * 10 + CUDA_BND_S_NEW]++;
  }
  thrust::copy(h_spine_cnts.begin(), h_spine_cnts.end(), d_spine_cnts.begin());
}
#endif

// ----------------------------------------------------------------------
// k_scan_scatter_received

static void __global__ k_scan_scatter_received(uint nr_recv, uint nr_prts_prev,
                                               uint* d_spine_sums,
                                               uint* d_bnd_off, uint* d_bidx,
                                               uint* d_ids)
{
  int n0 = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (n0 >= nr_recv) {
    return;
  }

  int n = n0 + nr_prts_prev;

  int nn = d_spine_sums[d_bidx[n] * 10 + CUDA_BND_S_NEW] + d_bnd_off[n0];
  d_ids[nn] = n;
}

// ----------------------------------------------------------------------
// scan_scatter_received

template <typename CudaMparticles>
void cuda_bndp<CudaMparticles, dim_yz>::scan_scatter_received(
  CudaMparticles* cmprts, uint n_prts_recv)
{
  if (n_prts_recv == 0) {
    return;
  }

  uint n_prts_prev = cmprts->n_prts - n_prts_recv;

  int dimGrid = (n_prts_recv + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

  k_scan_scatter_received<<<dimGrid, THREADS_PER_BLOCK>>>(
    n_prts_recv, n_prts_prev, d_spine_sums.data().get(), d_bnd_off.data().get(),
    cmprts->by_block_.d_idx.data().get(), cmprts->by_block_.d_id.data().get());
  cuda_sync_if_enabled();
}

// ----------------------------------------------------------------------
// scan_scatter_received_gold

template <typename CudaMparticles>
void cuda_bndp<CudaMparticles, dim_yz>::scan_scatter_received_gold(
  CudaMparticles* cmprts, uint n_prts_recv)
{
  thrust::host_vector<uint> h_bidx(cmprts->n_prts);
  thrust::host_vector<uint> h_bnd_off(n_prts_recv);
  thrust::host_vector<uint> h_id(cmprts->n_prts);
  thrust::host_vector<uint> h_spine_sums(1 + n_blocks * (10 + 1));

  thrust::copy(d_spine_sums.data(), d_spine_sums.data() + n_blocks * 11,
               h_spine_sums.begin());
  thrust::copy(cmprts->by_block_.d_idx.data(),
               cmprts->by_block_.d_idx.data() + cmprts->n_prts, h_bidx.begin());

  uint n_prts_prev = cmprts->n_prts - n_prts_recv;
  thrust::copy(d_bnd_off.begin(), d_bnd_off.end(), h_bnd_off.begin());
  for (int n0 = 0; n0 < n_prts_recv; n0++) {
    int n = n0 + n_prts_prev;
    int nn = h_spine_sums[h_bidx[n] * 10 + CUDA_BND_S_NEW] + h_bnd_off[n0];
    h_id[nn] = n;
  }
  thrust::copy(h_id.begin(), h_id.end(), cmprts->by_block_.d_id.begin());
}

// ----------------------------------------------------------------------
// sort_pairs_device

template <typename CudaMparticles>
void cuda_bndp<CudaMparticles, dim_yz>::sort_pairs_device(
  CudaMparticles* cmprts, uint n_prts_recv)
{
  static int pr_A, pr_B, pr_C, pr_D;
  if (!pr_B) {
    pr_A = prof_register("xchg_cnt_recvd", 1., 0, 0);
    pr_B = prof_register("xchg_top_scan", 1., 0, 0);
    pr_C = prof_register("xchg_ss_recvd", 1., 0, 0);
    pr_D = prof_register("xchg_bottom_scan", 1., 0, 0);
  }

  prof_start(pr_A);
  count_received(cmprts);
  prof_stop(pr_A);

  prof_start(pr_B);
  // FIXME why isn't 10 + 0 enough?
  thrust::exclusive_scan(d_spine_cnts.data(),
                         d_spine_cnts.data() + 1 + n_blocks * (10 + 1),
                         d_spine_sums.data());
  prof_stop(pr_B);

  prof_start(pr_C);
  scan_scatter_received(cmprts, n_prts_recv);
  prof_stop(pr_C);

  prof_start(pr_D);
  Int3 mx = b_mx();
  if (mx[0] == 1 && mx[1] == 4 && mx[2] == 4) {
    ScanScatterDigits3x<K, V, 0, RADIX_BITS, 0, NopFunctor<K>, NopFunctor<K>, 4,
                        4><<<n_blocks, B40C_RADIXSORT_THREADS>>>(
      d_spine_sums.data().get(), cmprts->by_block_.d_idx.data().get(),
      cmprts->by_block_.d_id.data().get(), cmprts->by_block_.d_off.data().get(),
      n_blocks);
  } else if (mx[0] == 1 && mx[1] == 8 && mx[2] == 8) {
    ScanScatterDigits3x<K, V, 0, RADIX_BITS, 0, NopFunctor<K>, NopFunctor<K>, 8,
                        8><<<n_blocks, B40C_RADIXSORT_THREADS>>>(
      d_spine_sums.data().get(), cmprts->by_block_.d_idx.data().get(),
      cmprts->by_block_.d_id.data().get(), cmprts->by_block_.d_off.data().get(),
      n_blocks);
  } else if (mx[0] == 1 && mx[1] == 16 && mx[2] == 16) {
    ScanScatterDigits3x<K, V, 0, RADIX_BITS, 0, NopFunctor<K>, NopFunctor<K>,
                        16, 16><<<n_blocks, B40C_RADIXSORT_THREADS>>>(
      d_spine_sums.data().get(), cmprts->by_block_.d_idx.data().get(),
      cmprts->by_block_.d_id.data().get(), cmprts->by_block_.d_off.data().get(),
      n_blocks);
  } else if (mx[0] == 1 && mx[1] == 32 && mx[2] == 32) {
    ScanScatterDigits3x<K, V, 0, RADIX_BITS, 0, NopFunctor<K>, NopFunctor<K>,
                        32, 32><<<n_blocks, B40C_RADIXSORT_THREADS>>>(
      d_spine_sums.data().get(), cmprts->by_block_.d_idx.data().get(),
      cmprts->by_block_.d_id.data().get(), cmprts->by_block_.d_off.data().get(),
      n_blocks);
  } else if (mx[0] == 1 && mx[1] == 64 && mx[2] == 64) {
    ScanScatterDigits3x<K, V, 0, RADIX_BITS, 0, NopFunctor<K>, NopFunctor<K>,
                        64, 64><<<n_blocks, B40C_RADIXSORT_THREADS>>>(
      d_spine_sums.data().get(), cmprts->by_block_.d_idx.data().get(),
      cmprts->by_block_.d_id.data().get(), cmprts->by_block_.d_off.data().get(),
      n_blocks);
  } else if (mx[0] == 1 && mx[1] == 128 && mx[2] == 128) {
    ScanScatterDigits3x<K, V, 0, RADIX_BITS, 0, NopFunctor<K>, NopFunctor<K>,
                        128, 128><<<n_blocks, B40C_RADIXSORT_THREADS>>>(
      d_spine_sums.data().get(), cmprts->by_block_.d_idx.data().get(),
      cmprts->by_block_.d_id.data().get(), cmprts->by_block_.d_off.data().get(),
      n_blocks);
  } else {
    printf("no support for b_mx %d x %d x %d!\n", mx[0], mx[1], mx[2]);
    assert(0);
  }
  cuda_sync_if_enabled();
  prof_stop(pr_D);

  // d_ids now contains the indices to reorder by
}

template <typename CudaMparticles>
void cuda_bndp<CudaMparticles, dim_yz>::sort_pairs_gold(CudaMparticles* cmprts,
                                                        uint n_prts_recv)
{
  thrust::host_vector<uint> h_bidx(cmprts->by_block_.d_idx.data(),
                                   cmprts->by_block_.d_idx.data() +
                                     cmprts->n_prts);
  thrust::host_vector<uint> h_id(cmprts->n_prts);
  thrust::host_vector<uint> h_off(cmprts->by_block_.d_off);
  thrust::host_vector<uint> h_spine_cnts(
    d_spine_cnts.data(), d_spine_cnts.data() + 1 + n_blocks * (10 + 1));

  thrust::host_vector<uint> h_spine_sums(1 + n_blocks * (10 + 1));

  for (int n = cmprts->n_prts - n_prts_recv; n < cmprts->n_prts; n++) {
    assert(h_bidx[n] < n_blocks);
    h_spine_cnts[h_bidx[n] * 10 + CUDA_BND_S_NEW]++;
  }

  thrust::exclusive_scan(h_spine_cnts.begin(), h_spine_cnts.end(),
                         h_spine_sums.begin());
  thrust::copy(h_spine_sums.begin(), h_spine_sums.end(), d_spine_sums.begin());

  Int3 mx = b_mx();
  for (int bid = 0; bid < n_blocks; bid++) {
    int b = bid % n_blocks_per_patch;
    int p = bid / n_blocks_per_patch;
    for (int n = h_off[bid]; n < h_off[bid + 1]; n++) {
      uint key = h_bidx[n];
      if (key < 9) {
        int dy = key % 3;
        int dz = key / 3;
        int by = b % mx[1];
        int bz = b / mx[1];
        uint bby = by + 1 - dy;
        uint bbz = bz + 1 - dz;
        assert(bby < mx[1] && bbz < mx[2]);
        uint bb = bbz * mx[1] + bby;
        int nn = h_spine_sums[(bb + p * n_blocks_per_patch) * 10 + key]++;
        h_id[nn] = n;
      } else { // OOB
        assert(0);
      }
    }
  }
  for (int n = cmprts->n_prts - n_prts_recv; n < cmprts->n_prts; n++) {
    int nn = h_spine_sums[h_bidx[n] * 10 + CUDA_BND_S_NEW]++;
    h_id[nn] = n;
  }

  thrust::copy(h_id.begin(), h_id.end(), cmprts->by_block_.d_id.begin());
  // d_ids now contains the indices to reorder by
}

template struct cuda_bndp<cuda_mparticles<BS144>, dim_yz>;

#endif
